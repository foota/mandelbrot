#include "hip/hip_runtime.h"
// madelbrot  by N. Futatsugi, 2020.06.18

#include <iostream>
#include <fstream>

#include <opencv2/opencv.hpp>

using namespace std;
using namespace cv;

const int BLOCK_SIZE_X = 16;
const int BLOCK_SIZE_Y = 16;

__device__
uchar4 coloring(int n)
{
	//const float ratio = 3.2f;
	const float ratio = 4.5f;
	const int H = 176;
	const int S = 128;
	const int V = 255;
	int c = static_cast<int>(n * ratio) / (V + 1);
	int x = static_cast<int>(n * ratio) % (V + 1);
	return make_uchar4(H, S, c % 2 == 0 ? V - x : x, 0);
}

__global__
void mandelbrot(float t, float l, float w, float h, int sw, int sh, int max_iter, float th, uchar4* d_color)
{
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int iy = blockIdx.y * blockDim.y + threadIdx.y;
	if (ix >= sw || iy >= sh) return;

	float ci = t + (static_cast<float>(iy) / sh) * h;
	float cr = l + (static_cast<float>(ix) / sw) * w;
	float zi = 0.0f;
	float zr = 0.0f;
	float zrzi, zr2, zi2;

	for (int i = 0; i < max_iter; i++) {
		zrzi = zr * zi;
		zr2 = zr * zr;
		zi2 = zi * zi;
		zr = zr2 - zi2 + cr;
		zi = zrzi + zrzi + ci;
		if (zi2 + zr2 >= th) {
			d_color[iy*sw+ix] = coloring(i);
			return;
		}
	}
	d_color[iy*sw+ix] = make_uchar4(0, 0, 0, 0);
}

void write_mandelbrot(const string outfile, float t, float l, float w, float h, int sw, int sh, int max_iter=256, float th=4.0f)
{
	dim3 num_blocks((sw - 1) / BLOCK_SIZE_X + 1, (sh - 1) / BLOCK_SIZE_Y + 1, 1);
	dim3 num_threads(BLOCK_SIZE_X, BLOCK_SIZE_Y, 1);

	uchar4* h_color;
	uchar4* d_color;
	hipHostMalloc(reinterpret_cast<void**>(&h_color), sizeof(uchar4) * sw * sh);
	hipMalloc(reinterpret_cast<void**>(&d_color), sizeof(uchar4) * sw * sh);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	
	mandelbrot<<<num_blocks, num_threads>>>(t, l, w, h, sw, sh, max_iter, th, d_color);
	hipDeviceSynchronize();

	hipEventRecord(stop);
	hipEventSynchronize(stop);

	hipMemcpy(h_color, d_color, sizeof(uchar4) * sw * sh, hipMemcpyDeviceToHost);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	cout << "Time (ms): " << milliseconds << endl;

	Mat h_color_8UC4(sh, sw, CV_8UC4, reinterpret_cast<unsigned char*>(&h_color[0]));
	Mat h_color_8UC3(sh, sw, CV_8UC3);
	cvtColor(h_color_8UC4, h_color_8UC3, COLOR_RGBA2RGB);
	cvtColor(h_color_8UC3, h_color_8UC3, COLOR_HSV2RGB);
	imwrite(outfile, h_color_8UC3);

	hipFree(d_color);
	hipHostFree(h_color);
}

int main(int argc, char* argv[])
{
	string outfile("mandelbrot_gpu.jpg");
	if (argc >= 2) outfile = argv[1];

	hipSetDevice(0);

	//write_mandelbrot(outfile, -1.0f, -2.0f, 2.666f, 2.0f, 640, 480);
	write_mandelbrot(outfile, 0.680f, -0.220f, 0.008f, 0.0045f, 2560*5, 1440*5, 10000, 10.0f);  // WQHD (16:9) xN

	return 0;
}
